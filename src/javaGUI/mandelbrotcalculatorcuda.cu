
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <time.h>
#include <string.h>

int xpixels = 1750;
int ypixels = 1000;
int currentPixel = 0;		//counter for calculations
int totalPixels;			//the total number of pixels in the image 

double xmin;			// ranges for the calculating the mandelbrot set on
double xmax;			//	real-imaginary plane, where x is real and y
double ymin;			//	is imaginary
double ymax;			//

char outputFile[] = "image.ppm";		// output file

/*
 * complex_number_t
 * Replresents a complex number by storing separate doubles for real and
 * imaginary parts for the complex number.
*/
typedef struct complex_number_t{
    double real;
    double imag;
}complex_number_t;

/*
 * pixel_t
 * Represents a pixel in the PPM file by storing the RGB components.
*/
typedef struct pixel_t{
    int r;
    int g;
    int b;
} pixel_t;

/*convert_to_2d
 *Finds index in a 1D array given 2D indices
 *INPUTS
 *	x - column position
 *	y - row position
 *	scale - number of elements per row
 *OUTPUT
 *	long index in the 1D array
*/
unsigned int convert_to_2d(int x, int y, int scale) {
    return y * scale + x;
}

/*
 *calculate_mandlebrot_set
 *Determines if a given position on the complex plane diverages or not and
 *outputs the results to a global array.
 *INPUTS
 *	x - horizontal position of pixel
 *	y - veritcal position of pixel
 *  xcoord - real component of complex number
 *	ycoord - imaginary component of complex number
 *	*pixelArray - global array representing output image
*/
__global__ void calculate_mandlebrot_set( pixel_t *pixelArray, int xpixels, int ypixels, double xmin, double xmax, double ymin, double ymax){
    int MAX_ITERATIONS = 1000;
    double xincrement = (xmax - xmin) / xpixels; // macro to find increments
    double yincrement = (ymax - ymin) / ypixels;

	complex_number_t z;
	z.real = 0;
	z.imag = 0;
	
	complex_number_t c;
	pixel_t *currPixel;

	if (threadIdx.x + blockDim.x * blockIdx.x < xpixels && threadIdx.y + blockDim.y * blockIdx.y < ypixels){
		int myIndex = (threadIdx.x + blockDim.x * blockIdx.x) +  xpixels* (threadIdx.y + blockDim.y * blockIdx.y);
		currPixel = pixelArray + myIndex;
		
		int x = threadIdx.x + blockDim.x * blockIdx.x;             // x posiiton of the pixel
		int y = threadIdx.y + blockDim.y * blockIdx.y;             // y position of the piel
		
		c.real = xmin + x * xincrement;
		c.imag = ymin + y * yincrement;
		
		int toBreak = 0;
		int i = 0;
		while( i < MAX_ITERATIONS ){
			complex_number_t temp;

			temp.real = z.real;
			temp.imag = z.imag;
			z.real = (z.real * z.real) - (z.imag * z.imag) + c.real; //real
			z.imag = (2*temp.real*temp.imag) + c.imag; // imaginary
			if(z.real*z.real + z.imag*z.imag > 4){
				//save colors to Pixel Array
				(*currPixel).r = (int) 255 * (((1.0*i) / (1.0 * MAX_ITERATIONS)));
				(*currPixel).g = (int) 255 * sqrt(((1.0*i) / (1.0 * MAX_ITERATIONS)));
				(*currPixel).b = (int) 50 * sqrt(1.0 - ((1.0*i) / (1.0 * MAX_ITERATIONS)));
				toBreak = 1;
				break; //end calculation for current pixel
			}
			i++;
		}
		// never escapes
		if(!toBreak){
			(*currPixel).r = 0;
			(*currPixel).g = 0;
			(*currPixel).b = 0;
		}
	}
}

/*
 *mandlebrot_thread
 *A worker thread that calculates if individual pixels are in the Mandelbrot set.
 *INPUTS
 *	*data - a pointer to and array of pixel_t for data output
*/
hipError_t calcMandelbrotCuda(pixel_t *data ){
	hipError_t cudaStatus;

    int d_xpixels, d_ypixels;
    double d_xmin, d_xmax, d_ymin, d_ymax;
	
	
	dim3 threadsPerBlock(32,32,1);
	dim3 numBlocks((xpixels - 1) / threadsPerBlock.x + 1, (ypixels - 1) / threadsPerBlock.y + 1, 1);
	
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?"); goto Error; }

    d_xpixels = xpixels;
    d_ypixels = ypixels;

    d_xmin = xmin;
    d_xmax = xmax;
    d_ymin = ymin;
    d_ymax = ymax;

	printf("threads.x = %d, threads.y = %d\n", threadsPerBlock.x, threadsPerBlock.y);
	printf("threads.x = %d, threads.y = %d\n", numBlocks.x, numBlocks.y);

	calculate_mandlebrot_set<<<numBlocks, threadsPerBlock>>>(data, d_xpixels, d_ypixels, d_xmin, d_xmax, d_ymin, d_ymax);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) { fprintf(stderr, "calculate_mandlebrot_set launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }
	
	hipDeviceSynchronize();

    Error:
    return cudaStatus;
}

int main(int argc, char * argv[]){
    if (!(argc == 5 || argc == 2)){
        printf("ERROR: incorrect number of arguments. Run with argument --help for help.\n");
        return 0;
    }
    
    if (argc == 2){
        // char *strHelp = "--help";
        if (!strcmp(argv[1], "--help")){
            printf("USAGE: CompiledMandelbrot [xmin] [xmax] [ymin] [ymax]\n");
        }
        else{
            printf("ERROR: Unrecognized argument. Run with argument --help for help.\n");
        }
        return 0;
    }
	    
	//set range of mandlebrot set from arguments
    xmin = atof(argv[1]);
    xmax = atof(argv[2]);
    ymin = atof(argv[3]);
    ymax = atof(argv[4]);
    
    if (ymin >= ymax){
        printf("ERROR: Invalid range for imaginary axis.\n");
        return 0;
    }
    
    if (xmin >= xmax){
        printf("ERROR: Invalid range for real axis.\n");
        return 0;
    }
    
	//initialize some values
	totalPixels	= xpixels * ypixels;
	
	//initialize output file with PPm header
    FILE *fp;
    fp = fopen(outputFile, "w+");
    fprintf(fp, "P3 \n%d %d \n255\n\n", xpixels, ypixels);
    fclose(fp);

    pixel_t *pixelArray; 

	//varaibles for timing
    struct timespec start, finish;
    double elapsed;

    printf("Timing Calculations...\n");
    clock_gettime(CLOCK_MONOTONIC, &start);

	
	//do cuda here
	hipMallocManaged(&pixelArray, totalPixels * sizeof(pixel_t));
	
	hipError_t cudaStatus = calcMandelbrotCuda(pixelArray);
    
    //calculate elapsed time
    clock_gettime(CLOCK_MONOTONIC, &finish);
    elapsed = (finish.tv_sec - start.tv_sec);
    elapsed += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;
    printf("Calculations took %f seconds", elapsed);

	//start file output timing
    printf("\nStarting file output...\n");
    clock_gettime(CLOCK_MONOTONIC, &start);
	
    fp = fopen(outputFile, "a");
    int x;
    int y;

    for (y = 0; y < ypixels; y++){
        for (x = 0; x < xpixels; x++){
            pixel_t* currPixel;
            currPixel = pixelArray + convert_to_2d(x, y, xpixels);		//get one pixel at a time
            fprintf(fp, " %d %d %d    ",(*currPixel).r,(*currPixel).g,(*currPixel).b);		//outpit pixel in PPM format
			
        }
        fprintf(fp, "\n");		//next line in PPM file

	}
    fclose(fp);
    //stop timing
    clock_gettime(CLOCK_MONOTONIC, &finish);
    elapsed = (finish.tv_sec - start.tv_sec);
    elapsed += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;
    printf("File output took %f seconds.\n", elapsed );
    
	
	//convert the PPM file to a PNG file to save space
    execl("/usr/bin/convert", "/usr/bin/convert", "image.ppm", "image.png", (char *)NULL);
    

}